
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>

using namespace std;

__global__ void reduceNeighbor(int* output, int* input) {
        int tid = threadIdx.x;
        int* inner_input = input + 8 * blockIdx.x * blockDim.x;
        inner_input[tid] += inner_input[tid + blockDim.x];
        inner_input[tid] += inner_input[tid + 2 * blockDim.x];
        inner_input[tid] += inner_input[tid + 3 * blockDim.x];
        inner_input[tid] += inner_input[tid + 4 * blockDim.x];
        inner_input[tid] += inner_input[tid + 5 * blockDim.x];
        inner_input[tid] += inner_input[tid + 6 * blockDim.x];
        inner_input[tid] += inner_input[tid + 7 * blockDim.x];
        __syncthreads();
        for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
                if (tid < stride) {
                        inner_input[tid] += inner_input[tid + stride];
                }
                __syncthreads();
        }
        if (tid < 32) {
                volatile int* vmem = inner_input;  // NOTE: very important 'volatile'
                vmem[tid] += vmem[tid + 32];
                vmem[tid] += vmem[tid + 16];
                vmem[tid] += vmem[tid + 8];
                vmem[tid] += vmem[tid + 4];
                vmem[tid] += vmem[tid + 2];
                vmem[tid] += vmem[tid + 1];
        }

        if (tid == 0) {
                output[blockIdx.x] = inner_input[0];
        }
}

static const int narr = (1 << 24);
static const int numThreads = 1024;
static const int numBlocks = (narr + numThreads - 1) / numThreads / 8;

struct TestResult {
        int sum;
        double elapsed;  // us
};

TestResult test(int* d_arr, int* h_arr, int* d_sum, int* h_sum) {
        hipMemcpy(d_arr, h_arr, narr * sizeof(int), hipMemcpyHostToDevice);
        dim3 grid(numBlocks);
        dim3 block(numThreads);
        auto start = std::chrono::high_resolution_clock::now();
        reduceNeighbor<<<grid, block>>>(d_sum, d_arr);
        hipDeviceSynchronize();
        hipMemcpy(h_sum, d_sum, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
        int sum = 0;
        for (int i = 0; i < numBlocks; ++i) {
                sum += h_sum[i];
        }
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        TestResult ret = { sum, elapsed.count() };
        return ret;
}

int main() {

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0); // 0是设备ID
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);

        srand(1);
        int* h_arr = new int[narr];
        for (int i = 0; i < narr; ++i) {
                h_arr[i] = (rand() & 0x7f);
        }

        int* d_arr = nullptr;
        hipMalloc(&d_arr, narr * sizeof(int));
        int* h_sum = new int[numBlocks];
        int* d_sum = nullptr;
        hipMalloc(&d_sum, numBlocks * sizeof(int));

        // warmup
        for (int i = 0; i < 100; ++i) {
                test(d_arr, h_arr, d_sum, h_sum);
        }

        // test
        TestResult result = test(d_arr, h_arr, d_sum, h_sum);
        cout << "gpu sum is: " << result.sum << endl << "elapsed time: " << result.elapsed * 1000 << " ms." << endl;

        delete[] h_arr;
        delete[] h_sum;
        hipFree(d_arr);
        hipFree(d_sum);
        return 0;
}
