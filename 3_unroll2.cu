
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>

using namespace std;

__global__ void reduceNeighbor(int* output, int* input) {
        int tid = threadIdx.x;
        int* inner_input = input + 2 * blockIdx.x * blockDim.x;
        int stride = blockDim.x;
        inner_input[tid] += inner_input[tid + stride];
        __syncthreads();
        stride >>= 1;
        for (; stride > 0; stride >>= 1) {
                if (tid < stride) {
                        inner_input[tid] += inner_input[tid + stride];
                }
                __syncthreads();
        }
        if (tid == 0) {
                output[blockIdx.x] = inner_input[0];
        }
}

static const int narr = (1 << 24);
static const int numThreads = 1024;
static const int numBlocks = (narr + numThreads - 1) / numThreads / 2;

struct TestResult {
        int sum;
        double elapsed;  // us
};

TestResult test(int* d_arr, int* h_arr, int* d_sum, int* h_sum) {
        hipMemcpy(d_arr, h_arr, narr * sizeof(int), hipMemcpyHostToDevice);
        dim3 grid(numBlocks);
        dim3 block(numThreads);
        auto start = std::chrono::high_resolution_clock::now();
        reduceNeighbor<<<grid, block>>>(d_sum, d_arr);
        hipDeviceSynchronize();
        hipMemcpy(h_sum, d_sum, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
        int sum = 0;
        for (int i = 0; i < numBlocks; ++i) {
                sum += h_sum[i];
        }
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        TestResult ret = { sum, elapsed.count() };
        return ret;
}

int main() {

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0); // 0是设备ID
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);

        srand(1);
        int* h_arr = new int[narr];
        for (int i = 0; i < narr; ++i) {
                h_arr[i] = (rand() & 0x7f);
        }

        int* d_arr = nullptr;
        hipMalloc(&d_arr, narr * sizeof(int));
        int* h_sum = new int[numBlocks];
        int* d_sum = nullptr;
        hipMalloc(&d_sum, numBlocks * sizeof(int));

        // warmup
        for (int i = 0; i < 100; ++i) {
                test(d_arr, h_arr, d_sum, h_sum);
        }

        // test
        TestResult result = test(d_arr, h_arr, d_sum, h_sum);
        cout << "gpu sum is: " << result.sum << endl << "elapsed time: " << result.elapsed * 1000 << " ms." << endl;

        delete[] h_arr;
        delete[] h_sum;
        hipFree(d_arr);
        hipFree(d_sum);
        return 0;
}
