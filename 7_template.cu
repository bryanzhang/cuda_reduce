
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>

using namespace std;

template <unsigned int iBlockSize>
__global__ void reduceNeighbor(int* output, int* input) {
        int tid = threadIdx.x;
        int* inner_input = input + 8 * blockIdx.x * blockDim.x;
        inner_input[tid] += inner_input[tid + blockDim.x];
        inner_input[tid] += inner_input[tid + 2 * blockDim.x];
        inner_input[tid] += inner_input[tid + 3 * blockDim.x];
        inner_input[tid] += inner_input[tid + 4 * blockDim.x];
        inner_input[tid] += inner_input[tid + 5 * blockDim.x];
        inner_input[tid] += inner_input[tid + 6 * blockDim.x];
        inner_input[tid] += inner_input[tid + 7 * blockDim.x];
        __syncthreads();

        if (iBlockSize >= 1024 && tid < 512) {
                inner_input[tid] += inner_input[tid + 512];
                __syncthreads();
        }

        if (iBlockSize >= 512 && tid < 256) {
                inner_input[tid] += inner_input[tid + 256];
                __syncthreads();
        }

        if (iBlockSize >= 256 && tid < 128) {
                inner_input[tid] += inner_input[tid + 128];
                __syncthreads();
        }

        if (iBlockSize >= 128 && tid < 64) {
                inner_input[tid] += inner_input[tid + 64];
                __syncthreads();
        }

        // unrolling warp
        if (tid < 32) {
                volatile int* vmem = inner_input;  // NOTE: very important 'volatile'
                vmem[tid] += vmem[tid + 32];
                vmem[tid] += vmem[tid + 16];
                vmem[tid] += vmem[tid + 8];
                vmem[tid] += vmem[tid + 4];
                vmem[tid] += vmem[tid + 2];
                vmem[tid] += vmem[tid + 1];
        }

        if (tid == 0) {
                output[blockIdx.x] = inner_input[0];
        }
}

static const int narr = (1 << 24);
static constexpr int numThreads = 1024;
static const int numBlocks = (narr + numThreads - 1) / numThreads / 8;

struct TestResult {
        int sum;
        double elapsed;  // us
};

TestResult test(int* d_arr, int* h_arr, int* d_sum, int* h_sum) {
        hipMemcpy(d_arr, h_arr, narr * sizeof(int), hipMemcpyHostToDevice);
        dim3 grid(numBlocks);
        dim3 block(numThreads);
        auto start = std::chrono::high_resolution_clock::now();
        reduceNeighbor<numThreads><<<grid, block>>>(d_sum, d_arr);
        hipDeviceSynchronize();
        hipMemcpy(h_sum, d_sum, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
        int sum = 0;
        for (int i = 0; i < numBlocks; ++i) {
                sum += h_sum[i];
        }
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        TestResult ret = { sum, elapsed.count() };
        return ret;
}

int main() {

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0); // 0是设备ID
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);

        srand(1);
        int* h_arr = new int[narr];
        for (int i = 0; i < narr; ++i) {
                h_arr[i] = (rand() & 0x7f);
        }

        int* d_arr = nullptr;
        hipMalloc(&d_arr, narr * sizeof(int));
        int* h_sum = new int[numBlocks];
        int* d_sum = nullptr;
        hipMalloc(&d_sum, numBlocks * sizeof(int));

        // warmup
        for (int i = 0; i < 100; ++i) {
                test(d_arr, h_arr, d_sum, h_sum);
        }

        // test
        TestResult result = test(d_arr, h_arr, d_sum, h_sum);
        cout << "gpu sum is: " << result.sum << endl << "elapsed time: " << result.elapsed * 1000 << " ms." << endl;

        delete[] h_arr;
        delete[] h_sum;
        hipFree(d_arr);
        hipFree(d_sum);
        return 0;
}
